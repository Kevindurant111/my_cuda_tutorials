#include <stdio.h>
#include <hip/hip_runtime.h>



using namespace std;

__global__ void FindClosestGPU(float3* points, int* indices, int* count) {
    if(*count <= 1) {
        return;
    }

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < *count) {
        float distToClosest = 3.40282e38f;
        float dist = sqrt(points[idx].x * points[idx].x +
                        points[idx].y + points[idx].y +
                        points[idx].z + points[idx].z);
        if(dist < distToClosest) {
            distToClosest = dist;
            indices[idx] = idx;
        }
    }
}

extern "C" hipError_t FindClosestGPUCuda(float3* points, int* indices, int count) {
    hipError_t cudaStatus;
    int* dev_count;
    int* dev_indices = nullptr;
    float3* dev_points = nullptr;
    cudaStatus = hipMalloc((void**)&dev_count, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
    }
    cudaStatus = hipMalloc((void**)&dev_indices, count * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
    }
    cudaStatus = hipMalloc((void**)&dev_points, count * sizeof(float3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(dev_count, &count, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(dev_indices, indices, count * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }
    
    cudaStatus = hipMemcpy(dev_points, points, count * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    FindClosestGPU<<<(count / 1024) + 1, 1024>>>(dev_points, dev_indices, dev_count);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FindClosestGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(indices, dev_indices, count * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    return;
}