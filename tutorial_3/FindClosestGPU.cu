#include <stdio.h>
#include <hip/hip_runtime.h>



using namespace std;
__device__ const int blockSize = 640;

__global__ void FindClosestGPU(float3* points, int* indices, int* count) {
    if(*count <= 1) {
        return;
    }

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < *count) {
        float distToClosest = 3.40282e38f;
        for(int i = 0; i < *count; i++) {
            if(i == idx) {
                continue;
            }
            float dist = sqrt((points[idx].x - points[i].x) * (points[idx].x - points[i].x) +
            (points[idx].y - points[i].y) * (points[idx].y - points[i].y) +
            (points[idx].z - points[i].z) * (points[idx].z - points[i].z));
            if(dist < distToClosest) {
                distToClosest = dist;
                indices[idx] = i;
            }
        }
    }
}

extern "C" hipError_t FindClosestGPUCuda(float3* points, int* indices, int count) {
    hipError_t cudaStatus;
    int* dev_count = nullptr;
    int* dev_indices = nullptr;
    float3* dev_points = nullptr;
    cudaStatus = hipMalloc((void**)&dev_count, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
    }
    cudaStatus = hipMalloc((void**)&dev_indices, count * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
    }
    cudaStatus = hipMalloc((void**)&dev_points, count * sizeof(float3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(dev_count, &count, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(dev_indices, indices, count * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }
    
    cudaStatus = hipMemcpy(dev_points, points, count * sizeof(float3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    FindClosestGPU<<<(count / blockSize) + 1, blockSize>>>(dev_points, dev_indices, dev_count);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FindClosestGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(indices, dev_indices, count * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }
}


__global__ void FindClosestGPUWithBlocking(float3* points, int* indices, int* count) {
    __shared__ float3 sharedPoints[blockSize];
    if(*count <= 1) {
        return;
    }

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    float3 thisPoint;
    float distToClosest = 3.40282e38f;
    if(idx < *count) {
        thisPoint = points[idx];

        for(int currentBlockOfPoints = 0; currentBlockOfPoints < gridDim.x; currentBlockOfPoints++) {
            if(threadIdx.x + currentBlockOfPoints * blockSize < *count) {
                sharedPoints[threadIdx.x] = points[threadIdx.x + currentBlockOfPoints * blockSize];
                __syncthreads();
            }
            
            for(int i = 0; i < blockSize; i++) {
                if(i + currentBlockOfPoints * blockSize == idx) {
                    continue;
                }
                float dist = sqrt((thisPoint.x - sharedPoints[i].x) * (thisPoint.x - sharedPoints[i].x) +
                (thisPoint.y - sharedPoints[i].y) * (thisPoint.y - sharedPoints[i].y) +
                (thisPoint.z - sharedPoints[i].z) * (thisPoint.z - sharedPoints[i].z));
                if((dist < distToClosest) && (i + currentBlockOfPoints * blockSize < *count)) {
                    distToClosest = dist;
                    indices[idx] = i + currentBlockOfPoints * blockSize;
                }
            }
            __syncthreads();
        }
    }
}

extern "C" hipError_t FindClosestGPUCudaWithBlocking(float3* points, int* indices, int count) {
    hipError_t cudaStatus;
    int* dev_count = nullptr;
    int* dev_indices = nullptr;
    float3* dev_points = nullptr;
    cudaStatus = hipMalloc((void**)&dev_count, sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
    }
    cudaStatus = hipMalloc((void**)&dev_indices, count * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
    }
    cudaStatus = hipMalloc((void**)&dev_points, count * sizeof(float3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(dev_count, &count, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(dev_indices, indices, count * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }
    
    cudaStatus = hipMemcpy(dev_points, points, count * sizeof(float3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    FindClosestGPUWithBlocking<<<(count / blockSize) + 1, blockSize>>>(dev_points, dev_indices, dev_count);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "FindClosestGPU launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }

    cudaStatus = hipMemcpy(indices, dev_indices, count * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_count);
        hipFree(dev_indices);
        hipFree(dev_points);
    }
}